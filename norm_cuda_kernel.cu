#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
template <typename scalar_t>
__global__ void norm_cuda_kernel(torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> weight,
                          torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> norm,
                          int n1, int n2,
                          int Wx,
                          int Wy,
                          int Nx,
                          int Ny )
{
  int startX = threadIdx.x * n1;
  int startY = threadIdx.y * n2;
  if(startX<Wx && startY<Wy){//valid thread
    scalar_t sum=0;
    scalar_t num=0;
    for(int i=0;i<n1;i++){
      for (int j=0;j<n2;j++){
        if(startX+i<Wx && startY+j<Wy){
          sum+=weight[startX+i][startY+j]*weight[startX+i][startY+j];
          num++;
        }
      }
    }
    norm[threadIdx.x][threadIdx.y]=sum/num;
  }
}

void norm_cuda(
    torch::Tensor weights,
    torch::Tensor out_norm,
    int n1,
    int n2)
{
  const auto WeightsSizeX = weights.size(0);
  const auto WeightsSizeY = weights.size(1);
  auto normSizeX=(WeightsSizeX+n1-1)/n1;
  auto normSizeY=(WeightsSizeY+n2-1)/n2;
  dim3 threadDim(8,8);
  dim3 blockDim((normSizeX+7/8),(normSizeY+7/8));
  
  

  AT_DISPATCH_FLOATING_TYPES(weights.type(), "norm_cuda", ([&] {
    norm_cuda_kernel<scalar_t><<<blockDim, threadDim>>>(
        weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        out_norm.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        n1,
        n2,
        WeightsSizeX,
        WeightsSizeY,
        normSizeX,
        normSizeY);
  }));
}